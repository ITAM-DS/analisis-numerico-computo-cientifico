
#include <hip/hip_runtime.h>
#include<stdio.h>
__global__ void suma(int a, int b, int *c){
    *c = a+b;
}
int main(void){
    int c;
    int *device_c;
    hipMalloc((void **)&device_c,sizeof(int));
    suma<<<1,1>>>(2,7,device_c);
    hipMemcpy(&c, device_c, sizeof(int), hipMemcpyDeviceToHost);
    printf("2+7 = %d\n", c);
    hipFree(device_c);
    return 0;
}
