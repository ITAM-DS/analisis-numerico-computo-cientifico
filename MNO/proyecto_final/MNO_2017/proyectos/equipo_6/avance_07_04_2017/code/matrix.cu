
//Multiplicacion de matriz usando un kernal compartido y usando uno no compartido


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#define TILE_WIDTH 2

/*multiplicacion de kernels de matriz*/

//no compartido
__global__ void
MatrixMul( float *Md , float *Nd , float *Pd , const int WIDTH )
{
           // calculate thread id
           unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;
           unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;

         for (int k = 0 ; k<WIDTH ; k++ )
         {
                  Pd[row*WIDTH + col]+= Md[row * WIDTH + k ] * Nd[ k * WIDTH + col];
          }
}

// compartido
__global__ void
MatrixMulSh( float *Md , float *Nd , float *Pd , const int WIDTH )
{

        //Taking shared array to break the MAtrix in Tile widht and fatch them in that array per ele
          __shared__ float Mds [TILE_WIDTH][TILE_WIDTH];
          __shared__ float Nds [TILE_WIDTH][TILE_WIDTH];

         // calculate thread id
          unsigned int col = TILE_WIDTH*blockIdx.x + threadIdx.x;
          unsigned int row = TILE_WIDTH*blockIdx.y + threadIdx.y;

        for (int m = 0 ; m<WIDTH/TILE_WIDTH ; m++ ) // m indicate number of phase
       {
            Mds[threadIdx.y][threadIdx.x] =  Md[row*WIDTH + (m*TILE_WIDTH + threadIdx.x)];
            Nds[threadIdx.y][threadIdx.x] =  Nd[ ( m*TILE_WIDTH + threadIdx.y) * WIDTH + col];
         __syncthreads() ; // for syncronizeing the threads

         // Do for tile
           for ( int k = 0; k<TILE_WIDTH ; k++ )
                       Pd[row*WIDTH + col]+= Mds[threadIdx.x][k] * Nds[k][threadIdx.y];
         __syncthreads() ; // for syncronizeing the threads

     }
}

// main routine
int main ()
{
   const int WIDTH = 6;
   float array1_h[WIDTH][WIDTH] ,array2_h[WIDTH][WIDTH],
                     result_array_h[WIDTH][WIDTH] ,M_result_array_h[WIDTH][WIDTH];
  float *array1_d, *array2_d, *result_array_d, *M_result_array_d; // device array
  int i , j ;
  //input in host array
  for ( i = 0 ; i<WIDTH ; i++ )
  {
     for (j = 0 ; j<WIDTH ; j++ )
     {
        array1_h[i][j] = 1;
        array2_h[i][j] = 2;
     }
  }

  //create device array cudaMalloc ( (void **)&array_name, sizeofmatrixinbytes) ;
  hipMalloc((void **) &array1_d, WIDTH*WIDTH*sizeof (int));
  hipMalloc((void **) &array2_d, WIDTH*WIDTH*sizeof (int));

 

  //copy host array to device array; cudaMemcpy ( dest , source , WIDTH , direction )
  hipMemcpy ( array1_d , array1_h , WIDTH*WIDTH*sizeof (int), hipMemcpyHostToDevice);
  hipMemcpy ( array2_d , array2_h , WIDTH*WIDTH*sizeof (int), hipMemcpyHostToDevice);

 

  //allocating memory for resultent device array
  hipMalloc((void **) &result_array_d , WIDTH*WIDTH*sizeof (int) );
  hipMalloc((void **) &M_result_array_d , WIDTH*WIDTH*sizeof (int) );

 

  //calling kernal
  dim3 dimGrid ( WIDTH/TILE_WIDTH , WIDTH/TILE_WIDTH ,1 );
  dim3 dimBlock( TILE_WIDTH, TILE_WIDTH, 1 );

// Change if 0 to if 1 for running non shared code and make if 0 for shared memory code
#if 0

                MatrixMul <<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH) ;

#endif
 
#if 1

               MatrixMulSh<<<dimGrid,dimBlock>>> ( array1_d , array2_d ,M_result_array_d , WIDTH);

#endif

  // all gpu function blocked till kernel is working
  //copy back result_array_d to result_array_h

  hipMemcpy(M_result_array_h , M_result_array_d , WIDTH*WIDTH*sizeof(int),
                                    hipMemcpyDeviceToHost);

  //printf the result array
  for ( i = 0 ; i<WIDTH ; i++ )
  {
      for ( j = 0 ; j < WIDTH ; j++ )
     {
        printf ("%f   ",M_result_array_h[i][j] );
     }
 printf ("\n");
}
 system("pause");
}
